#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include "sha256.cuh"

#define TOTAL_SIZE 108
#define MAX_SHARES 16

#define MIN(x, y) (((x) < (y)) ? (x) : (y))

__device__ __constant__ uint32_t prefix_c[TOTAL_SIZE/4];
__device__ __constant__ char share_chunk_c[64];
__device__ __constant__ size_t share_difficulty_c;

__device__ __forceinline__ void sha256_to_hex(unsigned char *hash, char *hex) {
    static const char digits[] = "0123456789abcdef";

    #pragma unroll
    for (int i = 0; i < 16; ++i) {
        char lo_nibble = digits[hash[i] & 0x0F];
        char hi_nibble = digits[(hash[i] & 0xF0) >> 4];
        *hex++ = hi_nibble;
        *hex++ = lo_nibble;
    }
    *hex = '\0';
}

__device__ __forceinline__ bool is_valid(const char* str) {
    int mask = 0;

    #pragma unroll
    for (int i = 0; i < share_difficulty_c; ++i) {
        mask |= (str[i] ^ share_chunk_c[i]);
    }
    return mask == 0;
}

__global__ void miner(unsigned char **out, int *stop, int *share_id) {
    const __restrict__ uint32_t tid = threadIdx.x;

    __shared__ SHA256_CTX prefix_ctx;
    if (tid == 0) {
        sha256_init(&prefix_ctx);
        sha256_update(&prefix_ctx, (unsigned char*)prefix_c, sizeof(uint32_t) * (TOTAL_SIZE-4)/4);
    }
    __syncthreads();

    uint32_t _hex[TOTAL_SIZE/4];
    memcpy(_hex, prefix_c, sizeof(uint32_t) * (TOTAL_SIZE-4)/4);

    SHA256_CTX ctx;
    unsigned char hash[32];
    char hash_hex[64];

    for (uint32_t index = blockIdx.x * blockDim.x + tid; *stop != 1; index += blockDim.x * gridDim.x) {
        _hex[TOTAL_SIZE/4-1] = index;

        memcpy(&ctx, &prefix_ctx, sizeof(SHA256_CTX));
        sha256_update(&ctx, (unsigned char*)&_hex[TOTAL_SIZE/4-1], 4);
        sha256_final(&ctx, hash);
        sha256_to_hex(hash, hash_hex);

        if (is_valid(hash_hex)) {
            int id = atomicAdd(share_id, 1);
            memcpy(out[id], _hex, sizeof(uint32_t) * TOTAL_SIZE/4);

            if (id >= MAX_SHARES-2) {
                *stop = 1;
            }
        }

        if (index >= 0xFFFFFFFF) {
            *stop = 1;
        }
    }
}

extern "C" {
    void start(const int device_id, const int threads, const int blocks, uint32_t *prefix, size_t difficulty, char *share_chunk, size_t share_difficulty, char *device_name, float *hashrate, unsigned char **out) {
        auto res = hipSetDevice(device_id);
        if (res != hipSuccess) {
            printf("Error setting device: %s\n", hipGetErrorString(res));
            return;
        }

        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, device_id);
        strcpy(device_name, deviceProp.name);

        checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(dev_k), host_k, sizeof(host_k), 0, hipMemcpyHostToDevice));

        // allocate memory on the device
        int *stop;
        hipMallocManaged(&stop, sizeof(int));
        hipMemcpy(stop, 0, sizeof(int), hipMemcpyHostToDevice);

        int *share_id;
        hipMallocManaged(&share_id, sizeof(int));
        hipMemcpy(share_id, 0, sizeof(int), hipMemcpyHostToDevice);

        unsigned char **out_g;
        unsigned char *out_t[MAX_SHARES];

        hipMalloc((void **)&out_g, MAX_SHARES*sizeof(unsigned char *));

        for (int i = 0; i < MAX_SHARES; ++i) {
            hipMalloc((void **)&out_t[i], sizeof(unsigned char) * TOTAL_SIZE);
        }
        hipMemcpy(out_g, out_t, sizeof(unsigned char *) * MAX_SHARES, hipMemcpyHostToDevice);

        for (int i = 0; i < MAX_SHARES; ++i) {
            hipMemcpy(out_t[i], out[i], sizeof(unsigned char) * TOTAL_SIZE, hipMemcpyHostToDevice);
        }

        hipError_t err;
        hipEvent_t start, end;
        float elapsed_ms = 0.0f;

        err = hipEventCreate(&start);
        if (err != hipSuccess) {
            printf("Failed to create start event: %s\n", hipGetErrorString(err));
        }

        err = hipEventCreate(&end);
        if (err != hipSuccess) {
            printf("Failed to create end event: %s\n", hipGetErrorString(err));
            hipEventDestroy(start);
        }

        err = hipEventRecord(start, 0);
        if (err != hipSuccess) {
            printf("Failed to record start event: %s\n", hipGetErrorString(err));
            hipEventDestroy(start);
            hipEventDestroy(end);
        }

        prefix[TOTAL_SIZE/4-2] = (prefix[TOTAL_SIZE/4-2] & 0xFFFF) | (difficulty << 16);

        hipMemcpyToSymbol(HIP_SYMBOL(share_chunk_c), share_chunk, sizeof(char) * 64);
        hipMemcpyToSymbol(HIP_SYMBOL(share_difficulty_c), &share_difficulty, sizeof(size_t));

        uint loops_count = 0;
        while (*share_id == 0 && loops_count < 5) {
            time_t now = time(NULL);
            prefix[TOTAL_SIZE/4-3] = (prefix[TOTAL_SIZE/4-3] & 0xFFFF) | ((now & 0xFFFF) << 16);
            prefix[TOTAL_SIZE/4-2] = (prefix[TOTAL_SIZE/4-2] & 0xFFFF0000) | ((now & 0xFFFF0000) >> 16);

            hipMemcpyToSymbol(HIP_SYMBOL(prefix_c), prefix, sizeof(uint32_t) * ((TOTAL_SIZE-4)/4));

            miner<<<threads,blocks>>> (out_g, stop, share_id);
            checkCudaErrors(hipDeviceSynchronize());

            *stop = 0;
            loops_count++;
        }

        err = hipEventRecord(end, 0);
        if (err != hipSuccess) {
            printf("Failed to record end event: %s\n", hipGetErrorString(err));
            hipEventDestroy(start);
            hipEventDestroy(end);
        }

        err = hipEventSynchronize(end);
        if (err != hipSuccess) {
            printf("Failed to synchronize end event: %s\n", hipGetErrorString(err));
            hipEventDestroy(start);
            hipEventDestroy(end);
        }

        err = hipEventElapsedTime(&elapsed_ms, start, end);
        if (err != hipSuccess) {
            printf("Failed to get elapsed time: %s\n", hipGetErrorString(err));
            hipEventDestroy(start);
            hipEventDestroy(end);
        }

        *hashrate = (4294967296.0 / (elapsed_ms / 1000.0) / 1000000000.0) * loops_count;

        if (*share_id > 0) {
            for (int i = 0; i < MIN(*share_id, MAX_SHARES); ++i) {
                hipMemcpy(out[i], out_t[i], sizeof(unsigned char) * TOTAL_SIZE, hipMemcpyDeviceToHost);
            }
        }

        for (int i = 0; i < MAX_SHARES; ++i) {
            hipFree(out_t[i]);
        }
        hipFree(out_g);

        hipFree(stop);
        hipFree(share_id);

        hipEventDestroy(start);
        hipEventDestroy(end);
    }
}
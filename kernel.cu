#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "sha256.cuh"

#define TOTAL_SIZE 108
#define MAX_SHARES 16

#define MIN(x, y) (((x) < (y)) ? (x) : (y))

__device__ void sha256_to_hex(unsigned char* data, char pout[64]) {
    const char* hex = "0123456789abcdef";
    for (int i = 0; i < 32; i++) {
        pout[i * 2] = hex[data[i] >> 4];
        pout[i * 2 + 1] = hex[data[i] & 0x0f];
    }
}

__device__ bool compare(const char* str_a, const char* str_b, unsigned len) {
    for (int i = 0; i < len; ++i) {
        if (str_a[i] != str_b[i])
            return false;
    }
    return true;
}

__global__ void miner(unsigned char *hash_prefix, char *share_chunk, size_t share_difficulty, unsigned char **out, int *stop, int *share_id) {
    uint32_t index = blockIdx.x * blockDim.x + threadIdx.x;

    unsigned char _hex[TOTAL_SIZE];
    for (int i = 0; i < TOTAL_SIZE-4; ++i) {
        _hex[i] = hash_prefix[i];
    }

    SHA256_CTX prefix_ctx;
    sha256_init(&prefix_ctx);
    sha256_update(&prefix_ctx, _hex, TOTAL_SIZE-4);

    while (*stop != 1) {
        _hex[TOTAL_SIZE-4] = index >> 24;
        _hex[TOTAL_SIZE-3] = index >> 16;
        _hex[TOTAL_SIZE-2] = index >> 8;
        _hex[TOTAL_SIZE-1] = index;

        SHA256_CTX ctx;
        memcpy(&ctx, &prefix_ctx, sizeof(SHA256_CTX));
        sha256_update(&ctx, _hex + (TOTAL_SIZE-4), 4);

        unsigned char hash[32];
        sha256_final(&ctx, hash);

        char hash_hex[64];
        sha256_to_hex(hash, hash_hex);

        if (compare(hash_hex, share_chunk, share_difficulty)) {
            memcpy(out[*share_id], _hex, sizeof(unsigned char) * TOTAL_SIZE);
            *share_id += 1;
        }
        if (index == 0xFFFFFFFF || *share_id == MAX_SHARES) {
            *stop = 1;
        }
        index += blockDim.x * gridDim.x;
    }
}

extern "C" {
    void start(const int device_id, const int threads, const int blocks, unsigned char *prefix, char *share_chunk, int share_difficulty, char *device_name, float *hashrate, unsigned char **out) {
        auto res = hipSetDevice(device_id);
        if (res != hipSuccess) {
            printf("Error setting device: %s\n", hipGetErrorString(res));
            return;
        }

        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, device_id);
        strcpy(device_name, deviceProp.name);

        checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(dev_k), host_k, sizeof(host_k), 0, hipMemcpyHostToDevice));

        // allocate memory on the device
        int *stop;
        hipMallocManaged(&stop, sizeof(int));
        hipMemcpy(stop, 0, sizeof(int), hipMemcpyHostToDevice);

        int *share_id;
        hipMallocManaged(&share_id, sizeof(int));
        hipMemcpy(share_id, 0, sizeof(int), hipMemcpyHostToDevice);

        char *share_chunk_g;
        hipMalloc(&share_chunk_g, sizeof(char) * share_difficulty);
        hipMemcpy(share_chunk_g, share_chunk, sizeof(char) * share_difficulty, hipMemcpyHostToDevice);

        unsigned char *prefix_g;
        hipMalloc(&prefix_g, sizeof(unsigned char) * (TOTAL_SIZE-4));
        hipMemcpy(prefix_g, prefix, sizeof(unsigned char) * (TOTAL_SIZE-4), hipMemcpyHostToDevice);

        unsigned char **out_g;
        unsigned char *out_t[MAX_SHARES];

        hipMalloc((void **)&out_g, MAX_SHARES*sizeof(unsigned char *));

        for (int i = 0; i < MAX_SHARES; ++i) {
            hipMalloc((void **)&out_t[i], sizeof(unsigned char) * TOTAL_SIZE);
        }
        hipMemcpy(out_g, out_t, sizeof(unsigned char *) * MAX_SHARES, hipMemcpyHostToDevice);

        for (int i = 0; i < MAX_SHARES; ++i) {
            hipMemcpy(out_t[i], out[i], sizeof(unsigned char) * TOTAL_SIZE, hipMemcpyHostToDevice);
        }

        hipError_t err;
        hipEvent_t start, end;
        float elapsed_ms = 0.0f;

        err = hipEventCreate(&start);
        if (err != hipSuccess) {
            printf("Failed to create start event: %s\n", hipGetErrorString(err));
        }

        err = hipEventCreate(&end);
        if (err != hipSuccess) {
            printf("Failed to create end event: %s\n", hipGetErrorString(err));
            hipEventDestroy(start);
        }

        err = hipEventRecord(start, 0);
        if (err != hipSuccess) {
            printf("Failed to record start event: %s\n", hipGetErrorString(err));
            hipEventDestroy(start);
            hipEventDestroy(end);
        }

        miner<<<threads,blocks>>> (prefix_g, share_chunk_g, share_difficulty, out_g, stop, share_id);
        checkCudaErrors(hipDeviceSynchronize());

        err = hipEventRecord(end, 0);
        if (err != hipSuccess) {
            printf("Failed to record end event: %s\n", hipGetErrorString(err));
            hipEventDestroy(start);
            hipEventDestroy(end);
        }

        err = hipEventSynchronize(end);
        if (err != hipSuccess) {
            printf("Failed to synchronize end event: %s\n", hipGetErrorString(err));
            hipEventDestroy(start);
            hipEventDestroy(end);
        }

        err = hipEventElapsedTime(&elapsed_ms, start, end);
        if (err != hipSuccess) {
            printf("Failed to get elapsed time: %s\n", hipGetErrorString(err));
            hipEventDestroy(start);
            hipEventDestroy(end);
        }

        *hashrate = 4294967296.0 / (elapsed_ms / 1000.0) / 1000000.0;

        if (*share_id > 0) {
            for (int i = 0; i < MIN(*share_id, MAX_SHARES); ++i) {
                hipMemcpy(out[i], out_t[i], sizeof(unsigned char) * TOTAL_SIZE, hipMemcpyDeviceToHost);
            }
        }

        for (int i = 0; i < MAX_SHARES; ++i) {
            hipFree(out_t[i]);
        }
        hipFree(out_g);

        hipFree(stop);
        hipFree(share_id);
        hipFree(share_chunk_g);
        hipFree(prefix_g);

        hipEventDestroy(start);
        hipEventDestroy(end);
    }
}
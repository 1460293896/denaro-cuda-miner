#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include "sha256.cuh"

#define TOTAL_SIZE 108
#define MAX_SHARES 16
#define UNROLL_FACTOR 4

#define MIN(x, y) (((x) < (y)) ? (x) : (y))

__device__ void sha256_to_hex(unsigned char *hash, char *hex) {
    static const char *digits = "0123456789abcdef";
    for (int i = 0; i < 16; i++) {
        char lo_nibble = digits[hash[i] & 0x0F];
        char hi_nibble = digits[(hash[i] & 0xF0) >> 4];
        *hex++ = hi_nibble;
        *hex++ = lo_nibble;
    }
    *hex = '\0';
}

__device__ bool compare(const char* str_a, const char* str_b, size_t len) {
    for (size_t i = 0; i < len; ++i) {
        if (str_a[i] != str_b[i]) {
            return false;
        }
    }
    return true;
}

__global__ void miner(uint32_t *prefix, char *share_chunk, size_t share_difficulty, unsigned char **out, int *stop, int *share_id) {
    uint32_t tid = threadIdx.x;
    uint32_t index = blockIdx.x * blockDim.x + tid;

    __shared__ SHA256_CTX prefix_ctx;
    __shared__ uint32_t prefix_copy[TOTAL_SIZE/4 - 1];
    if (tid == 0) {
        memcpy(prefix_copy, prefix, sizeof(uint32_t) * (TOTAL_SIZE-4)/4);
        sha256_init(&prefix_ctx);
        sha256_update(&prefix_ctx, (unsigned char*)prefix_copy, (TOTAL_SIZE-4));
    }
    __syncthreads();

    uint32_t _hex[TOTAL_SIZE/4];
    memcpy(_hex, prefix_copy, sizeof(uint32_t) * (TOTAL_SIZE-4)/4);

    SHA256_CTX ctx;
    unsigned char hash[32];
    char hash_hex[64];

    uint32_t i = 0;
    while (*stop != 1) {
        _hex[TOTAL_SIZE/4-1] = index + i * blockDim.x * gridDim.x;

        memcpy(&ctx, &prefix_ctx, sizeof(SHA256_CTX));
        sha256_update(&ctx, (unsigned char*)&_hex[TOTAL_SIZE/4-1], 4);
        sha256_final(&ctx, hash);
        sha256_to_hex(hash, hash_hex);

        if (compare(hash_hex, share_chunk, share_difficulty)) {
            int id = atomicAdd(share_id, 1);
            memcpy(out[id], _hex, sizeof(uint32_t) * TOTAL_SIZE/4);

            if (id >= MAX_SHARES-2) {
                *stop = 1;
            }
        }

        i++;

        if (i == UNROLL_FACTOR) {
            index += i * blockDim.x * gridDim.x;
            i = 0;
        }

        if (index >= 0xFFFFFFFF) {
            *stop = 1;
        }
    }
}

extern "C" {
    void start(const int device_id, const int threads, const int blocks, uint32_t *prefix, char *share_chunk, int share_difficulty, char *device_name, float *hashrate, unsigned char **out) {
        auto res = hipSetDevice(device_id);
        if (res != hipSuccess) {
            printf("Error setting device: %s\n", hipGetErrorString(res));
            return;
        }

        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, device_id);
        strcpy(device_name, deviceProp.name);

        checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(dev_k), host_k, sizeof(host_k), 0, hipMemcpyHostToDevice));

        // allocate memory on the device
        int *stop;
        hipMallocManaged(&stop, sizeof(int));
        hipMemcpy(stop, 0, sizeof(int), hipMemcpyHostToDevice);

        int *share_id;
        hipMallocManaged(&share_id, sizeof(int));
        hipMemcpy(share_id, 0, sizeof(int), hipMemcpyHostToDevice);

        char *share_chunk_g;
        hipMalloc(&share_chunk_g, sizeof(char) * share_difficulty);
        hipMemcpy(share_chunk_g, share_chunk, sizeof(char) * share_difficulty, hipMemcpyHostToDevice);

        uint32_t *prefix_g;
        hipMalloc(&prefix_g, sizeof(uint32_t) * ((TOTAL_SIZE-4)/4));
        hipMemcpy(prefix_g, prefix, sizeof(uint32_t) * ((TOTAL_SIZE-4)/4), hipMemcpyHostToDevice);

        unsigned char **out_g;
        unsigned char *out_t[MAX_SHARES];

        hipMalloc((void **)&out_g, MAX_SHARES*sizeof(unsigned char *));

        for (int i = 0; i < MAX_SHARES; ++i) {
            hipMalloc((void **)&out_t[i], sizeof(unsigned char) * TOTAL_SIZE);
        }
        hipMemcpy(out_g, out_t, sizeof(unsigned char *) * MAX_SHARES, hipMemcpyHostToDevice);

        for (int i = 0; i < MAX_SHARES; ++i) {
            hipMemcpy(out_t[i], out[i], sizeof(unsigned char) * TOTAL_SIZE, hipMemcpyHostToDevice);
        }

        hipError_t err;
        hipEvent_t start, end;
        float elapsed_ms = 0.0f;

        err = hipEventCreate(&start);
        if (err != hipSuccess) {
            printf("Failed to create start event: %s\n", hipGetErrorString(err));
        }

        err = hipEventCreate(&end);
        if (err != hipSuccess) {
            printf("Failed to create end event: %s\n", hipGetErrorString(err));
            hipEventDestroy(start);
        }

        err = hipEventRecord(start, 0);
        if (err != hipSuccess) {
            printf("Failed to record start event: %s\n", hipGetErrorString(err));
            hipEventDestroy(start);
            hipEventDestroy(end);
        }

        miner<<<threads,blocks>>> (prefix_g, share_chunk_g, share_difficulty, out_g, stop, share_id);
        checkCudaErrors(hipDeviceSynchronize());

        err = hipEventRecord(end, 0);
        if (err != hipSuccess) {
            printf("Failed to record end event: %s\n", hipGetErrorString(err));
            hipEventDestroy(start);
            hipEventDestroy(end);
        }

        err = hipEventSynchronize(end);
        if (err != hipSuccess) {
            printf("Failed to synchronize end event: %s\n", hipGetErrorString(err));
            hipEventDestroy(start);
            hipEventDestroy(end);
        }

        err = hipEventElapsedTime(&elapsed_ms, start, end);
        if (err != hipSuccess) {
            printf("Failed to get elapsed time: %s\n", hipGetErrorString(err));
            hipEventDestroy(start);
            hipEventDestroy(end);
        }

        *hashrate = 4294967296.0 / (elapsed_ms / 1000.0) / 1000000000.0;

        if (*share_id > 0) {
            for (int i = 0; i < MIN(*share_id, MAX_SHARES); ++i) {
                hipMemcpy(out[i], out_t[i], sizeof(unsigned char) * TOTAL_SIZE, hipMemcpyDeviceToHost);
            }
        }

        for (int i = 0; i < MAX_SHARES; ++i) {
            hipFree(out_t[i]);
        }
        hipFree(out_g);

        hipFree(stop);
        hipFree(share_id);
        hipFree(share_chunk_g);
        hipFree(prefix_g);

        hipEventDestroy(start);
        hipEventDestroy(end);
    }
}